#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
//
// The MIT License
//
// Copyright (c) 2006 Scientific Computing and Imaging Institute,
// University of Utah (USA)
//
// License for the specific language governing rights and limitations under
// Permission is hereby granted, free of charge, to any person obtaining a
// copy of this software and associated documentation files (the "Software"),
// to deal in the Software without restriction, including without limitation
// the rights to use, copy, modify, merge, publish, distribute, sublicense,
// and/or sell copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included
// in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
// OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
// THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
// DEALINGS IN THE SOFTWARE.
//
///////////////////////////////////////////////////////////////////////////////

#ifndef ELVIS_CUT_SURFACE_CONTOUR_MODULE_CU
#define ELVIS_CUT_SURFACE_CONTOUR_MODULE_CU

#include <optix_cuda.h>
#include <optix_math.h>
#include <optixu/optixu_matrix.h>
#include <optixu/optixu_aabb.h>
#include <ElVis/Core/CutSurfacePayloads.cu>
#include <ElVis/Core/ConvertToColor.cu>
#include <ElVis/Core/Float.h>
#include <ElVis/Core/Interval.hpp>
#include <ElVis/Core/IntervalPoint.cu>
#include <ElVis/Core/ElementId.h>

rtBuffer<ElVisFloat, 2> ContourSampleBuffer;
//rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<ElVisFloat3, 2> ReferencePointAtIntersectionBuffer;
rtBuffer<unsigned int, 2> ElementIdAtIntersectionBuffer;
rtBuffer<unsigned int, 2> ElementTypeAtIntersectionBuffer;

rtBuffer<ElVisFloat, 1> Isovalues;

rtDeclareVariable(int, TreatElementBoundariesAsDiscontinuous, , );

RT_PROGRAM void ContourMiss()
{
}

__device__ __forceinline__ ElVisFloat3 CalculateRayDirection(const uint2& pixelIndex, const ElVisFloat2& offset)
{
    ElVisFloat2 screen = MakeFloat2(color_buffer.size());
    ElVisFloat2 pixelSize = MAKE_FLOAT(2.0)/screen;
    
    ElVisFloat2 pixelOffset = offset/MAKE_FLOAT(2.0);
    
    ElVisFloat2 d = MakeFloat2(launch_index) / screen * MAKE_FLOAT(2.0) - MAKE_FLOAT(1.0);
    d = d + pixelSize * pixelOffset;
    
    //float3 ray_origin = eye;
    ElVisFloat3 ray_direction = normalize(d.x*U + d.y*V + W);
    return ray_direction;
}

// raw_color_buffer 
RT_PROGRAM void SamplePixelCornersRayGeneratorForCategorization()
{
    ELVIS_PRINTF("SamplePixelCornersRayGenerator\n");
    // Note - there are occlusion issues here.
    ElVisFloat2 screen = MakeFloat2(color_buffer.size());
    ElVisFloat2 pixelSize = MAKE_FLOAT(2.0)/screen;
    
    ElVisFloat x = MAKE_FLOAT(-1.0);
    ElVisFloat y = MAKE_FLOAT(-1.0);
    ElVisFloat2 pixelOffset = MakeFloat2(x, y)*MAKE_FLOAT(.5);
    
    ElVisFloat2 d = MakeFloat2(launch_index) / screen * MAKE_FLOAT(2.0) - MAKE_FLOAT(1.0);
    d = d + pixelSize * pixelOffset;
    
    ElVisFloat3 ray_origin = eye;
    ElVisFloat3 ray_direction = CalculateRayDirection(launch_index, MakeFloat2(MAKE_FLOAT(-1.0), MAKE_FLOAT(-1.0)));//normalize(d.x*U + d.y*V + W);

    optix::Ray ray = optix::make_Ray(ConvertToFloat3(ray_origin), ConvertToFloat3(ray_direction), 0, 1e-3f, RT_DEFAULT_MAX);
    CutSurfaceScalarValuePayload payload;
    
    payload.Initialize();
    payload.isValid = false;
    payload.scalarValue = ELVIS_FLOAT_MAX;
    payload.Normal = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
    payload.Color = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));
    rtTrace(SurfaceGeometryGroup, ray, payload);
    
    ELVIS_PRINTF("SamplePixelCornersRayGeneratorForCategorization: Scalar Value (%f)\n", payload.scalarValue);
    ELVIS_PRINTF("SamplePixelCornersRayGeneratorForCategorization: Element Id (%d)\n", payload.elementId);
    ELVIS_PRINTF("SamplePixelCornersRayGeneratorForCategorization: Element Type (%d)\n", payload.elementType);
    ELVIS_PRINTF("SamplePixelCornersRayGeneratorForCategorization: Reference Point (%f, %f, %f)\n", payload.ReferenceIntersectionPoint.x,
                 payload.ReferenceIntersectionPoint.y, payload.ReferenceIntersectionPoint.z);
    ContourSampleBuffer[launch_index] = payload.scalarValue;
    ReferencePointAtIntersectionBuffer[launch_index] = payload.ReferenceIntersectionPoint;
    ElementIdAtIntersectionBuffer[launch_index] = payload.elementId;
    ElementTypeAtIntersectionBuffer[launch_index] = payload.elementType;

}

__device__ __forceinline__ ElVis::Interval<ElVisFloat> EvaluateBetweenReferencePoints(unsigned int elementId, unsigned int elementType, const ElVisFloat3& p0, const ElVisFloat3& p1)
{
     ElVis::Interval<ElVisFloat> r0(fminf(p0.x, p1.x), fmaxf(p0.x, p1.x));
     ElVis::Interval<ElVisFloat> s0(fminf(p0.y, p1.y), fmaxf(p0.y, p1.y));
     ElVis::Interval<ElVisFloat> t0(fminf(p0.z, p1.z), fmaxf(p0.z, p1.z));

     IntervalPoint ip(r0, s0, t0);
     ElVis::Interval<ElVisFloat> result;
     SampleScalarFieldAtReferencePointOptiX(elementId, elementType, FieldId, ip, ip,
                                            result);
     return result;
}

__device__ __forceinline__ ElVis::Interval<ElVisFloat> EvaluateEdge(unsigned int elementId, unsigned int elementType, uint2 i0, uint2 i1)
{
     ElVisFloat3 p0 = ReferencePointAtIntersectionBuffer[i0];
     ElVisFloat3 p1 = ReferencePointAtIntersectionBuffer[i1];

     ELVIS_PRINTF("EvaluateEdge: P0 = (%f, %f, %f), P1 = (%f, %f, %f).\n",
                  p0.x, p0.y, p0.z, p1.x, p1.y, p1.z);
     return EvaluateBetweenReferencePoints(elementId, elementType, p0, p1);
}

// p0 - The reference point at the beginning of the interval.
// p1 - The reference point at the end of the interval.
__device__ __forceinline__ ElVis::Interval<ElVisFloat> SubdivideInterval1(unsigned int elementId, unsigned int elementType,
                                  const ElVisFloat3& p0, const ElVisFloat3& p1, 
                                  const ElVisFloat2& cornerOffset0, const ElVisFloat2& cornerOffset1)
{
    // Input is two pixel corners.  We'll do up to two levels of subdivision to start with and see how that affects
    // the final image.
    // First level is a ray at 1/2 between pixel corners.
    // Seconds level is at 1/4 and 3/4.

    ElVisFloat2 offset;
    if( cornerOffset0.x == cornerOffset1.x )
    {
        // Vertical edge
        offset = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(0.0));
    }
    else
    {
        // horizontal edge.
        offset = MakeFloat2(MAKE_FLOAT(0.0), cornerOffset0.y);
    }

    ElVisFloat3 ray_direction = CalculateRayDirection(launch_index, offset); 

    ElVisFloat3 ray_origin = eye;
    optix::Ray ray = optix::make_Ray(ConvertToFloat3(ray_origin), ConvertToFloat3(ray_direction), 2, 1e-3f, RT_DEFAULT_MAX);
    CutSurfaceScalarValuePayload payload;
    payload.Initialize();
    payload.isValid = false;
    payload.scalarValue = ELVIS_FLOAT_MAX;
    payload.Normal = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
    payload.Color = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));
    rtTrace(SurfaceGeometryGroup, ray, payload);
    
    // Now evaluate the interval math between p0-mid and mid-p1 to see if we can reject
    // this pixel.
    ElVisFloat3 mid = payload.ReferenceIntersectionPoint;
    ElVis::Interval<ElVisFloat> i0 = EvaluateBetweenReferencePoints(elementId, elementType, p0, mid);
    ElVis::Interval<ElVisFloat> i1 = EvaluateBetweenReferencePoints(elementId, elementType, mid, p1);

    return ElVis::Interval<ElVisFloat>(fminf(i0.GetLow(), i1.GetLow()), fmaxf(i0.GetHigh(), i1.GetHigh()));
}

__device__ __forceinline__ ElVis::Interval<ElVisFloat> SubdivideInterval2(unsigned int elementId, unsigned int elementType,
                                  const ElVisFloat3& p0, const ElVisFloat3& p1, 
                                  const ElVisFloat2& cornerOffset0, const ElVisFloat2& cornerOffset1)
{
    // Input is two pixel corners.  We'll do up to two levels of subdivision to start with and see how that affects
    // the final image.
    // First level is a ray at 1/2 between pixel corners.
    // Seconds level is at 1/4 and 3/4.

    ElVisFloat2 offset[3];
    if( cornerOffset0.x == cornerOffset1.x )
    {
        // Vertical edge
        offset[0] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(-.5));
        offset[1] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(0.0));
        offset[2] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(.5));
    }
    else
    {
        offset[0] = MakeFloat2(MAKE_FLOAT(-.5), cornerOffset0.y);
        offset[1] = MakeFloat2(MAKE_FLOAT(0.0), cornerOffset0.y);
        offset[2] = MakeFloat2(MAKE_FLOAT(.5), cornerOffset0.y);
    }

    ElVisFloat3 ray_direction[] = {
        CalculateRayDirection(launch_index, offset[0]),
        CalculateRayDirection(launch_index, offset[1]),
        CalculateRayDirection(launch_index, offset[2]) }; 

    CutSurfaceScalarValuePayload payload[3];
    ElVisFloat3 ray_origin = eye;

    for(unsigned int i = 0; i < 3; ++i)
    {
        optix::Ray ray = optix::make_Ray(ConvertToFloat3(ray_origin), ConvertToFloat3(ray_direction[i]), 2, 1e-3f, RT_DEFAULT_MAX);

        payload[i].Initialize();
        payload[i].isValid = false;
        payload[i].scalarValue = ELVIS_FLOAT_MAX;
        payload[i].Normal = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        payload[i].Color = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));
        rtTrace(SurfaceGeometryGroup, ray, payload[i]);
    }

    // Now evaluate the interval math between p0-mid and mid-p1 to see if we can reject
    // this pixel.
    ElVis::Interval<ElVisFloat> i0 = EvaluateBetweenReferencePoints(elementId, elementType, p0, payload[0].ReferenceIntersectionPoint);
    ElVis::Interval<ElVisFloat> i1 = EvaluateBetweenReferencePoints(elementId, elementType, payload[0].ReferenceIntersectionPoint, payload[1].ReferenceIntersectionPoint);
    ElVis::Interval<ElVisFloat> i2 = EvaluateBetweenReferencePoints(elementId, elementType, payload[1].ReferenceIntersectionPoint, payload[2].ReferenceIntersectionPoint);
    ElVis::Interval<ElVisFloat> i3 = EvaluateBetweenReferencePoints(elementId, elementType, payload[2].ReferenceIntersectionPoint, p1);

    i0.Combine(i1);
    i0.Combine(i2);
    i0.Combine(i3);
    return i0;
}

__device__ __forceinline__ ElVis::Interval<ElVisFloat> SubdivideInterval3(unsigned int elementId, unsigned int elementType,
                                  const ElVisFloat3& p0, const ElVisFloat3& p1, 
                                  const ElVisFloat2& cornerOffset0, const ElVisFloat2& cornerOffset1)
{
    // Input is two pixel corners.  We'll do up to two levels of subdivision to start with and see how that affects
    // the final image.
    // First level is a ray at 1/2 between pixel corners.
    // Seconds level is at 1/4 and 3/4.

    ElVisFloat2 offset[7];
    if( cornerOffset0.x == cornerOffset1.x )
    {
        // Vertical edge
        offset[0] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(-.75));
        offset[1] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(-.5));
        offset[2] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(-.25));
        offset[3] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(0.0));
        offset[4] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(.25));
        offset[5] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(.5));
        offset[6] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(.75));
    }
    else
    {
        offset[0] = MakeFloat2(MAKE_FLOAT(-.75), cornerOffset0.y);
        offset[1] = MakeFloat2(MAKE_FLOAT(-.5), cornerOffset0.y);
        offset[2] = MakeFloat2(MAKE_FLOAT(-.25), cornerOffset0.y);
        offset[3] = MakeFloat2(MAKE_FLOAT(0.0), cornerOffset0.y);
        offset[4] = MakeFloat2(MAKE_FLOAT(.25), cornerOffset0.y);
        offset[5] = MakeFloat2(MAKE_FLOAT(.5), cornerOffset0.y);
        offset[6] = MakeFloat2(MAKE_FLOAT(.75), cornerOffset0.y);
    }

    CutSurfaceScalarValuePayload payload[7];
    ElVisFloat3 ray_origin = eye;

    for(unsigned int i = 0; i < 7; ++i)
    {
        ElVisFloat3 ray_direction = CalculateRayDirection(launch_index, offset[i]);
        optix::Ray ray = optix::make_Ray(ConvertToFloat3(ray_origin), ConvertToFloat3(ray_direction), 2, 1e-3f, RT_DEFAULT_MAX);

        payload[i].Initialize();
        payload[i].isValid = false;
        payload[i].scalarValue = ELVIS_FLOAT_MAX;
        payload[i].Normal = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        payload[i].Color = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));
        rtTrace(SurfaceGeometryGroup, ray, payload[i]);
    }

    // Now evaluate the interval math between p0-mid and mid-p1 to see if we can reject
    // this pixel.
    ElVis::Interval<ElVisFloat> result = EvaluateBetweenReferencePoints(elementId, elementType, p0, payload[0].ReferenceIntersectionPoint);
    for(unsigned int i = 0; i < 6; ++i)
    {
        ElVis::Interval<ElVisFloat> i1 = EvaluateBetweenReferencePoints(elementId, elementType, payload[i].ReferenceIntersectionPoint, payload[i+1].ReferenceIntersectionPoint);
        result.Combine(i1);
    }
    ElVis::Interval<ElVisFloat> i3 = EvaluateBetweenReferencePoints(elementId, elementType, payload[6].ReferenceIntersectionPoint, p1);
    result.Combine(i3);
    return result;
}

RT_PROGRAM void CategorizeMeshPixels()
{
    // Corner testing.
    // c0 = lower left corner
    // c1 = lower right corner
    // c2 = upper left corner
    // c3 = upper right corner
    uint2 c0_index = make_uint2(launch_index.x, launch_index.y);
    uint2 c1_index = make_uint2(launch_index.x, launch_index.y);;
    c1_index.x += 1;
    uint2 c2_index = make_uint2(launch_index.x, launch_index.y);;
    c2_index.y += 1;
    uint2 c3_index = make_uint2(launch_index.x, launch_index.y);;
    c3_index.x += 1;
    c3_index.y += 1;

    ElVis::ElementId id0;
    ElVis::ElementId id1;
    ElVis::ElementId id2;
    ElVis::ElementId id3;

    id0.Id = ElementIdAtIntersectionBuffer[c0_index];
    id1.Id = ElementIdAtIntersectionBuffer[c1_index];
    id2.Id = ElementIdAtIntersectionBuffer[c2_index];
    id3.Id = ElementIdAtIntersectionBuffer[c3_index];

    id0.Type = ElementTypeAtIntersectionBuffer[c0_index];
    id1.Type = ElementTypeAtIntersectionBuffer[c1_index];
    id2.Type = ElementTypeAtIntersectionBuffer[c2_index];
    id3.Type = ElementTypeAtIntersectionBuffer[c3_index];

    if( id0.Id == -1 ||
        id1.Id == -1 ||
        id2.Id == -1 ||
        id3.Id == -1 )
    {
        return;
    }

    bool pixelIsElementBoundary =
     ( id0 != id1 ||
        id1 != id2 ||
        id2 != id3 );

    if( pixelIsElementBoundary )
    {
        raw_color_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.5), MAKE_FLOAT(0.5), MAKE_FLOAT(0.5));
        color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
        normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        SampleBuffer[launch_index] = ELVIS_FLOAT_MAX;
    }
}

rtDeclareVariable(int, MatchVisual3Contours, , );

RT_PROGRAM void CategorizeContourPixels()
{
    ELVIS_PRINTF("CategorizeContourPixels\n");
    // Corner testing.
    // c0 = lower left corner
    // c1 = lower right corner
    // c2 = upper left corner
    // c3 = upper right corner
    uint2 c0_index = make_uint2(launch_index.x, launch_index.y);
    uint2 c1_index = make_uint2(launch_index.x, launch_index.y);;
    c1_index.x += 1;
    uint2 c2_index = make_uint2(launch_index.x, launch_index.y);;
    c2_index.y += 1;
    uint2 c3_index = make_uint2(launch_index.x, launch_index.y);;
    c3_index.x += 1;
    c3_index.y += 1;
    
    ElVisFloat c0 = ContourSampleBuffer[c0_index];
    ElVisFloat c1 = ContourSampleBuffer[c1_index];
    ElVisFloat c2 = ContourSampleBuffer[c2_index];
    ElVisFloat c3 = ContourSampleBuffer[c3_index];
    
    // The 5000 are to get around a bug for the demos but needs to be fixed.
    bool allSamplesValid = true;
    allSamplesValid = (c0 != ELVIS_FLOAT_MAX) &&
        (c1 != ELVIS_FLOAT_MAX) &&
        (c2 != ELVIS_FLOAT_MAX) &&
        (c3 != ELVIS_FLOAT_MAX) &&
        c0 < 5000 && c1 < 5000 && c2 < 5000 && c3 < 5000;


    ElVisFloat3 visual3BackgroundColor = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));

    ELVIS_PRINTF("CategorizeContourPixels: All Samples Valid %d\n", allSamplesValid);
    if( !allSamplesValid ) 
    {
        if( MatchVisual3Contours )
        {
            raw_color_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));;
            color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
            SampleBuffer[launch_index] = ELVIS_FLOAT_MAX;
            normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        }
        return;
    }

    ElVis::ElementId id0;
    ElVis::ElementId id1;
    ElVis::ElementId id2;
    ElVis::ElementId id3;


    id0.Id = ElementIdAtIntersectionBuffer[c0_index];
    id1.Id = ElementIdAtIntersectionBuffer[c1_index];
    id2.Id = ElementIdAtIntersectionBuffer[c2_index];
    id3.Id = ElementIdAtIntersectionBuffer[c3_index];

    id0.Type = ElementTypeAtIntersectionBuffer[c0_index];
    id1.Type = ElementTypeAtIntersectionBuffer[c1_index];
    id2.Type = ElementTypeAtIntersectionBuffer[c2_index];
    id3.Type = ElementTypeAtIntersectionBuffer[c3_index];


    bool pixelIsElementBoundary =
     ( id0 != id1 ||
        id1 != id2 ||
        id2 != id3 );

    if( TreatElementBoundariesAsDiscontinuous && pixelIsElementBoundary )
    {
        if( MatchVisual3Contours )
        {
            raw_color_buffer[launch_index] = visual3BackgroundColor;
            color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
            SampleBuffer[launch_index] = ELVIS_FLOAT_MAX;
            normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        }
        return;
    }

    ElVisFloat3 elementBoundaryColor = MakeFloat3(MAKE_FLOAT(.5), MAKE_FLOAT(.5), MAKE_FLOAT(.5));
    ElVisFloat3 contourColor = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
    ElVisFloat3 ambiguousColor = MakeFloat3(MAKE_FLOAT(.25), MAKE_FLOAT(.5), MAKE_FLOAT(.5));
    ElVisFloat3 secondLevelAmbiguous = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(.5), MAKE_FLOAT(.5));


    bool oneIsovalueIsValid = false;
    if( MatchVisual3Contours )
    {
        bool oneIsovalueIsValid = false;
        int numIsovalues = Isovalues.size();
        ELVIS_PRINTF("CategorizeContourPixels: Num isovalues: %d\n", numIsovalues);
        for(int isoValueIndex = 0; isoValueIndex < Isovalues.size(); ++isoValueIndex)
        {
            ElVisFloat isovalue = Isovalues[isoValueIndex];
            ELVIS_PRINTF("CategorizeContourPixels: testing isovalue: %f\n", isovalue);
            bool lowerThanOneValue = (isovalue <= c0) || (isovalue <= c1) || (isovalue <= c2) || (isovalue <= c3);
            bool higherThanOneValue = (isovalue >= c0) || (isovalue >= c1) || (isovalue >= c2) || (isovalue >= c3);

            if( lowerThanOneValue && higherThanOneValue )
            {
                oneIsovalueIsValid = true;
                break;

            }
        }

        if( !oneIsovalueIsValid )
        {
            raw_color_buffer[launch_index] = visual3BackgroundColor;
            color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
            SampleBuffer[launch_index] = ELVIS_FLOAT_MAX;
            normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        }
    }
    else
    {
        int numIsovalues = Isovalues.size();
        ELVIS_PRINTF("CategorizeContourPixels: Num isovalues: %d\n", numIsovalues);
        for(int isoValueIndex = 0; isoValueIndex < Isovalues.size(); ++isoValueIndex)
        {
            ElVisFloat isovalue = Isovalues[isoValueIndex];
            ELVIS_PRINTF("CategorizeContourPixels: testing isovalue: %f\n", isovalue);
            bool lowerThanOneValue = (isovalue <= c0) || (isovalue <= c1) || (isovalue <= c2) || (isovalue <= c3);
            bool higherThanOneValue = (isovalue >= c0) || (isovalue >= c1) || (isovalue >= c2) || (isovalue >= c3);

            if( lowerThanOneValue && higherThanOneValue )
            {
                ELVIS_PRINTF("(%d, %d), Isovalue %f and corners %f, %f, %f, %f\n", launch_index.x, launch_index.y, isovalue, c0, c1, c2, c3);
                // BLACK - contour
                raw_color_buffer[launch_index] = contourColor;
                color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
                normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));

                // Clear out the sample buffer to prevent the color mapper from kicking in.
                SampleBuffer[launch_index] = ELVIS_FLOAT_MAX;
                oneIsovalueIsValid = true;
            }
        }
    }

    if( !oneIsovalueIsValid )
    {
        ELVIS_PRINTF("CategorizeContourPixels: No isovalues valid.\n");
        // If the element types are all different, then I can't use interval arithmetic.
        if( ElementIdAtIntersectionBuffer[c0_index] != ElementIdAtIntersectionBuffer[c1_index] ||
            ElementIdAtIntersectionBuffer[c0_index] != ElementIdAtIntersectionBuffer[c2_index] ||
            ElementIdAtIntersectionBuffer[c0_index] != ElementIdAtIntersectionBuffer[c3_index] ||
            ElementTypeAtIntersectionBuffer[c0_index] != ElementTypeAtIntersectionBuffer[c1_index] || 
            ElementTypeAtIntersectionBuffer[c0_index] != ElementTypeAtIntersectionBuffer[c2_index] ||
            ElementTypeAtIntersectionBuffer[c0_index] != ElementTypeAtIntersectionBuffer[c3_index])
        {            
            // Mark the element boundary.
            raw_color_buffer[launch_index] = elementBoundaryColor;
            color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
            normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
            return;
        }

        // No Subdivisions
        unsigned int id = ElementIdAtIntersectionBuffer[c0_index];
        unsigned int type = ElementTypeAtIntersectionBuffer[c0_index];

        ElVis::Interval<ElVisFloat> edge0 = EvaluateEdge(id, type, c0_index, c1_index);
        ElVis::Interval<ElVisFloat> edge1 = EvaluateEdge(id, type, c0_index, c2_index);
        ElVis::Interval<ElVisFloat> edge2 = EvaluateEdge(id, type, c1_index, c3_index);
        ElVis::Interval<ElVisFloat> edge3 = EvaluateEdge(id, type, c2_index, c3_index);

        bool mayContainAnIsovalue = false;
        for(int isoValueIndex = 0; isoValueIndex < Isovalues.size(); ++isoValueIndex)
        {
            float isovalue = Isovalues[isoValueIndex];
            ELVIS_PRINTF("CategorizeContourPixels: Testing ambiguous %f: (%f, %f), (%f, %f), (%f, %f), (%f, %f).\n",
                         isovalue, edge0.GetLow(), edge0.GetHigh(),
                         edge1.GetLow(), edge1.GetHigh(),
                         edge2.GetLow(), edge2.GetHigh(),
                         edge3.GetLow(), edge3.GetHigh());

            if(edge0.Contains(isovalue) || edge1.Contains(isovalue) ||
                edge2.Contains(isovalue) || edge3.Contains(isovalue) )
            {
                ELVIS_PRINTF("CategorizeContourPixels: May contain isovalue %f: (%f, %f), (%f, %f), (%f, %f), (%f, %f).\n",
                             isovalue, edge0.GetLow(), edge0.GetHigh(),
                             edge1.GetLow(), edge1.GetHigh(),
                             edge2.GetLow(), edge2.GetHigh(),
                             edge3.GetLow(), edge3.GetHigh());
                mayContainAnIsovalue = true;
            }
        }

        if( !mayContainAnIsovalue )
        {
            // Definitely not: WHITE
            raw_color_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));
            color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
            normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
            return;
        }

        const int numSubdivisions = 3;
        if( numSubdivisions == 1 )
        {
            edge0 = SubdivideInterval1(id, type,
                    ReferencePointAtIntersectionBuffer[c0_index],
                    ReferencePointAtIntersectionBuffer[c2_index],
                    make_float2(-1.0, -1.0), make_float2(-1.0, 1.0));
            edge1 = SubdivideInterval1(id, type,
                ReferencePointAtIntersectionBuffer[c1_index],
                ReferencePointAtIntersectionBuffer[c3_index],
                make_float2(1.0, -1.0), make_float2(1.0, 1.0));

            edge2 = SubdivideInterval1(id, type,
                ReferencePointAtIntersectionBuffer[c0_index],
                ReferencePointAtIntersectionBuffer[c1_index],
                make_float2(-1.0, -1.0), make_float2(1.0, -1.0));

            edge3 = SubdivideInterval1(id, type,
                ReferencePointAtIntersectionBuffer[c2_index],
                ReferencePointAtIntersectionBuffer[c3_index],
                make_float2(-1.0, 1.0), make_float2(1.0, 1.0));
        }

        if( numSubdivisions == 2 )
        {
            edge0 = SubdivideInterval2(id, type,
                    ReferencePointAtIntersectionBuffer[c0_index],
                    ReferencePointAtIntersectionBuffer[c2_index],
                    make_float2(-1.0, -1.0), make_float2(-1.0, 1.0));
            edge1 = SubdivideInterval2(id, type,
                ReferencePointAtIntersectionBuffer[c1_index],
                ReferencePointAtIntersectionBuffer[c3_index],
                make_float2(1.0, -1.0), make_float2(1.0, 1.0));

            edge2 = SubdivideInterval2(id, type,
                ReferencePointAtIntersectionBuffer[c0_index],
                ReferencePointAtIntersectionBuffer[c1_index],
                make_float2(-1.0, -1.0), make_float2(1.0, -1.0));

            edge3 = SubdivideInterval2(id, type,
                ReferencePointAtIntersectionBuffer[c2_index],
                ReferencePointAtIntersectionBuffer[c3_index],
                make_float2(-1.0, 1.0), make_float2(1.0, 1.0));
        }

        if( numSubdivisions == 3 )
        {
            edge0 = SubdivideInterval3(id, type,
                    ReferencePointAtIntersectionBuffer[c0_index],
                    ReferencePointAtIntersectionBuffer[c2_index],
                    make_float2(-1.0, -1.0), make_float2(-1.0, 1.0));
            edge1 = SubdivideInterval3(id, type,
                ReferencePointAtIntersectionBuffer[c1_index],
                ReferencePointAtIntersectionBuffer[c3_index],
                make_float2(1.0, -1.0), make_float2(1.0, 1.0));

            edge2 = SubdivideInterval3(id, type,
                ReferencePointAtIntersectionBuffer[c0_index],
                ReferencePointAtIntersectionBuffer[c1_index],
                make_float2(-1.0, -1.0), make_float2(1.0, -1.0));

            edge3 = SubdivideInterval3(id, type,
                ReferencePointAtIntersectionBuffer[c2_index],
                ReferencePointAtIntersectionBuffer[c3_index],
                make_float2(-1.0, 1.0), make_float2(1.0, 1.0));
        }

        mayContainAnIsovalue = false;
        for(int isoValueIndex = 0; isoValueIndex < Isovalues.size(); ++isoValueIndex)
        {
            float isovalue = Isovalues[isoValueIndex];
            if(edge0.Contains(isovalue) || edge1.Contains(isovalue) ||
                edge2.Contains(isovalue) || edge3.Contains(isovalue) )
            {
                mayContainAnIsovalue = true;
            }
        }

        if( mayContainAnIsovalue )
        {
            raw_color_buffer[launch_index] = ambiguousColor;
            color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
            normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        }
        else
        {
            // Definitely not: WHITE
            raw_color_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));
            color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
            normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        }
    }
}

#endif
