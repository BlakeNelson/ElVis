#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
//
// The MIT License
//
// Copyright (c) 2006 Scientific Computing and Imaging Institute,
// University of Utah (USA)
//
// License for the specific language governing rights and limitations under
// Permission is hereby granted, free of charge, to any person obtaining a
// copy of this software and associated documentation files (the "Software"),
// to deal in the Software without restriction, including without limitation
// the rights to use, copy, modify, merge, publish, distribute, sublicense,
// and/or sell copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included
// in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
// OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
// THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
// DEALINGS IN THE SOFTWARE.
//
///////////////////////////////////////////////////////////////////////////////

#ifndef ELVIS_EXTENSIONS_NEKTAR_PLUS_PLUS_EXTENSION_OPTIX_QUADRILATERAL_CU
#define ELVIS_EXTENSIONS_NEKTAR_PLUS_PLUS_EXTENSION_OPTIX_QUADRILATERAL_CU

#include <ElVis/Extensions/NektarPlusPlusExtension/Expansions.cu>

__device__ __forceinline__ ElVisFloat EvaluateQuadAtReferencePoint(
    ElVisFloat *coeffs, uint2 *modes, const ElVisFloat2& p)
{
    int cnt = 0;
    ElVisFloat result = MAKE_FLOAT(0.0);

    for(unsigned int j = 0; j < modes->y; ++j)
    {
        ElVisFloat value_j = ModifiedA(j, p.y);
        for(unsigned int i = 0; i < modes->x; ++i)
        {
            result += coeffs[cnt++] * ModifiedA(i, p.x) * value_j;
        }
    }

    return result;
}

__device__ __forceinline__ ElVisFloat EvaluateQuadGradientAtReferencePoint0(
    ElVisFloat *coeffs, uint2 *modes, const ElVisFloat2& p)
{
    int cnt = 0;
    ElVisFloat result = MAKE_FLOAT(0.0);

    for(unsigned int j = 0; j < modes->y; ++j)
    {
        ElVisFloat value_j = ModifiedA(j, p.y);
        for(unsigned int i = 0; i < modes->x; ++i)
        {
            result += coeffs[cnt++] * ModifiedAPrime(i, p.x) * value_j;
        }
    }

    return result;
}

__device__ __forceinline__ ElVisFloat EvaluateQuadGradientAtReferencePoint1(
    ElVisFloat *coeffs, uint2 *modes, const ElVisFloat2& p)
{
    int cnt = 0;
    ElVisFloat result = MAKE_FLOAT(0.0);

    for(unsigned int j = 0; j < modes->y; ++j)
    {
        ElVisFloat value_j = ModifiedAPrime(j, p.y);
        for(unsigned int i = 0; i < modes->x; ++i)
        {
            result += coeffs[cnt++] * ModifiedA(i, p.x) * value_j;
        }
    }

    return result;
}

#endif
