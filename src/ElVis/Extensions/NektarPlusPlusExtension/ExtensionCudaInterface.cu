#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
//
// The MIT License
//
// Copyright (c) 2006 Scientific Computing and Imaging Institute,
// University of Utah (USA)
//
// License for the specific language governing rights and limitations under
// Permission is hereby granted, free of charge, to any person obtaining a
// copy of this software and associated documentation files (the "Software"),
// to deal in the Software without restriction, including without limitation
// the rights to use, copy, modify, merge, publish, distribute, sublicense,
// and/or sell copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included
// in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
// OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
// THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
// DEALINGS IN THE SOFTWARE.
//
///////////////////////////////////////////////////////////////////////////////

#ifndef ELVIS_NEKTAR_PLUS_PLUS_EXTENSION_EXTENSION_INTERFACE_CU
#define ELVIS_NEKTAR_PLUS_PLUS_EXTENSION_EXTENSION_INTERFACE_CU

#include <ElVis/Extensions/NektarPlusPlusExtension/typedefs.cu>
#include <ElVis/Core/Float.cu>
#include <ElVis/Core/typedefs.cu>
#include <ElVis/Core/Interval.hpp>
#include <ElVis/Extensions/NektarPlusPlusExtension/CudaHexahedron.cu>
#include <ElVis/Core/IntervalPoint.cu>

__device__ ElVisFloat4* FaceVertexBuffer;
__device__ ElVisFloat4* FaceNormalBuffer;



ELVIS_DEVICE void EstimateRangeCuda(unsigned int elementId, unsigned int elementType, int fieldId,
                                          const ElVisFloat3& p0, const ElVisFloat3& p1,
                                          ElVis::Interval<ElVisFloat>& result)
{
//    TensorPoint t0 = ConvertToTensorSpaceCuda(elementId, elementType, p0);
//    TensorPoint t1 = ConvertToTensorSpaceCuda(elementId, elementType, p1);
//    IntervalPoint interval(t0, t1);
//    result = EvaluateFieldAtTensorPointCuda(elementId, elementType, fieldId, interval.x, interval.y, interval.z);
////    if( elementType == 0 )
////    {
////        ElVisFloat3 t0 = TransformWorldToTensor(elementId, p0);
////        ElVisFloat3 t1 = TransformWorldToTensor(elementId, p1);

////        IntervalPoint interval(t0, t1);
////        result = EvaluateHexFieldAtTensorPoint<ElVis::Interval<ElVisFloat> >(elementId, interval);
////    }
////    else if( elementType == 1 )
////    {
////        ElVisFloat3 t0 = TransformPrismWorldToTensor(PrismVertexBuffer, elementId, p0);
////        ElVisFloat3 t1 = TransformPrismWorldToTensor(PrismVertexBuffer, elementId, p1);

////        IntervalPoint interval(t0, t1);
////        result = EvaluatePrismFieldAtTensorPoint<ElVis::Interval<ElVisFloat> >(elementId, interval);
////    }
}

ELVIS_DEVICE void CalculateTransposedInvertedMappingJacobianCuda(unsigned int elementId, unsigned int elementType, int fieldId, const TensorPoint& tp, ElVisFloat* J)
{
    ElVisFloat JInv[9];

    if( elementType == 0 )
    {
        calculateInverseJacobian(elementId, tp, JInv);
    }

    J[0] = JInv[0];
    J[1] = JInv[3];
    J[2] = JInv[6];

    J[3] = JInv[1];
    J[4] = JInv[4];
    J[5] = JInv[7];

    J[6] = JInv[2];
    J[7] = JInv[5];
    J[8] = JInv[8];
}

ELVIS_DEVICE ElVisFloat3 CalculateTensorGradient(unsigned int elementId, unsigned int elementType, int fieldId, const TensorPoint& p)
{
    ElVisFloat3 result = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
    //if( elementType == 0 )
    //{
    //    uint3 degree = HexDegrees[elementId];

    //    uint coefficientIndex = HexCoefficientIndices[elementId];
    //    ElVisFloat* coeffs = &(HexCoefficients[coefficientIndex]);

    //    result.x = EvaluateHexGradientDir1AtTensorPoint(degree, p.x, p.y, p.z, coeffs);
    //    result.y = EvaluateHexGradientDir2AtTensorPoint(degree, p.x, p.y, p.z, coeffs);
    //    result.z = EvaluateHexGradientDir3AtTensorPoint(degree, p.x, p.y, p.z, coeffs);
    //}
    //else if( elementType == 1 )
    //{
    //    uint3 degree = PrismDegrees[elementId];

    //    uint coefficientIndex = PrismCoefficientIndices[elementId];
    //    ElVisFloat* coeffs = &(PrismCoefficients[coefficientIndex]);

    //    result.x = EvaluatePrismGradientDir1AtTensorPoint<ElVisFloat>(degree, p.x, p.y, p.z, coeffs);
    //    result.y = EvaluatePrismGradientDir2AtTensorPoint<ElVisFloat>(degree, p.x, p.y, p.z, coeffs);
    //    result.z = EvaluatePrismGradientDir3AtTensorPoint<ElVisFloat>(degree, p.x, p.y, p.z, coeffs);
    //}
    return result;
}







ELVIS_DEVICE ElVisError ConvertWorldToReferenceSpaceCuda(int elementId, int elementType, const WorldPoint& wp,
                                                          ElVis::ReferencePointParameterType referenceType, ReferencePoint& result)
{
    ElVisError returnVal = eNoError;
    if( elementType == 0 )
    {
        result = TransformNektarPlusPlusHexWorldToTensorCuda(elementId, wp);
    }
    else
    {
        returnVal = eInvalidElementType;
    }
    return returnVal;
}


template<typename PointType, typename ResultType>
ELVIS_DEVICE ElVisError SampleScalarFieldAtReferencePointCuda(int elementId, int elementType, int fieldId,
                                                               const PointType& worldPoint,
                                                               const PointType& tp,
                                                               ResultType& result)
{
    ElVisError returnVal = eNoError;
    if( elementType == 0 )
    {
        result = EvaluateNektarPlusPlusHexAtTensorPointCuda(elementId, tp);
    }
    else
    {
        returnVal = eInvalidElementType;
    }
    return returnVal;
}


ELVIS_DEVICE ElVisFloat3 EvaluateNormalCuda(unsigned int elementId, unsigned int elementType, int fieldId, const ElVisFloat3& worldPoint)
{
    ElVisFloat3 result = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));

    ReferencePoint tp;
    ConvertWorldToReferenceSpaceCuda(elementId, elementType, worldPoint, ElVis::eReferencePointIsInvalid, tp);

    ElVisFloat3 tv = CalculateTensorGradient(elementId, elementType, fieldId, tp);

    ElVisFloat J[9];
    CalculateTransposedInvertedMappingJacobianCuda(elementId, elementType, fieldId, tp, J);

    result.x = tv.x*J[0] + tv.y*J[1] + tv.z*J[2];
    result.y = tv.x*J[3] + tv.y*J[4] + tv.z*J[5];
    result.z = tv.x*J[6] + tv.y*J[7] + tv.z*J[8];

//    ELVIS_PRINTF("Normal Vector %f, %f, %f\n", result.x, result.y, result.z);
    return result;
}


#endif
